
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) {
    printf("Thread %d says: Hello World from GPU!\n",threadIdx.x);
}

int main(void) {
    printf("Hello World from CPU!\n");
    helloFromGPU <<<1,10>>>();
    hipDeviceReset();
    // cudaDeviceSynchronize();
    return 0;
}