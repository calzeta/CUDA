#include "hip/hip_runtime.h"


#include <cstdio>

__global__ void printHelloGPU()
{
	printf("Hello World from the GPU\n");
}

int main()
{
	printHelloGPU<<<5,5>>>();
	hipDeviceSynchronize();
	
	getchar();
	return 0;
}